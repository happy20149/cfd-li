#include "hip/hip_runtime.h"
#pragma once
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "Utilities.hpp"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define at(AR, I, J) AR[imax * (J) + (I)]

__device__ Real convection_u(Real U[6], Real V[6], Real inv_dx, Real inv_dy, Real gamma) {
    Real result = 0.0;
    Real interp1 = (U[0] + U[1]) / 2;
    Real interp2 = (U[2] + U[1]) / 2;
    Real interp3 = interp1 * 2;
    Real interp4 = (U[1] - U[0]) / 2;
    Real interp5 = interp2 * 2;
    Real interp6 = (U[2] - U[1]) / 2;

    Real interp7 = (V[0] + V[1]) / 2;
    Real interp8 = (U[3] + U[1]) / 2;
    Real interp9 = (V[5] + V[4]) / 2;
    Real interp10 = (U[4] + U[1]) / 2;
    Real interp11 = interp7 * 2;
    Real interp12 = (U[1] - U[3]) / 2;
    Real interp13 = interp9 * 2;
    Real interp14 = (U[4] - U[1]) / 2;

    // dU^2/dx
    Real result_fd = (interp1 * interp1 - interp2 * interp2) * inv_dx;
    Real result_dc = gamma * (abs(interp3) / 2 * interp4 - abs(interp5) / 2 * interp6) * inv_dx;
    result += result_fd + result_dc;

    // dUV/dy
    result_fd = (interp7 * interp8 - interp9 * interp10) * inv_dy;
    result_dc = gamma * (abs(interp11) / 2 * interp12 - abs(interp13) / 2 * interp14) * inv_dy;
    result += result_fd + result_dc;

    return result;
}

__device__ Real convection_uT(Real U[2], Real T[5], Real inv_dx, Real gamma) {
    Real interp1 = (T[1] + T[0]) / 2;
    Real interp2 = (T[2] + T[1]) / 2;
    Real interp3 = (T[1] - T[0]) / 2;
    Real interp4 = (T[2] - T[1]) / 2;

    return inv_dx * ((U[1] * interp1 - U[0] * interp2) + gamma * (abs(U[1]) * interp3 - abs(U[0]) * interp4));
}

__device__ Real convection_vT(Real V[2], Real T[5], Real inv_dy, Real gamma) {
    Real result = 0.0;

    Real interp1 = (T[1] + T[3]) / 2;
    Real interp2 = (T[4] + T[1]) / 2;
    Real interp3 = (T[1] - T[3]) / 2;
    Real interp4 = (T[4] - T[1]) / 2;

    return inv_dy * ((V[1] * interp1 - V[0] * interp2) + gamma * (abs(V[1]) * interp3 - abs(V[0]) * interp4));
}

__device__ Real convection_v(Real U[6], Real V[6], Real inv_dx, Real inv_dy, Real gamma) {
    Real result = 0.0;
    Real interp1 = (V[3] + V[1]) / 2;
    Real interp2 = (V[4] + V[1]) / 2;
    Real interp3 = interp1 * 2;
    Real interp4 = (V[1] - V[3]) / 2;
    Real interp5 = interp2 * 2;
    Real interp6 = (V[4] - V[1]) / 2;

    Real interp7 = (U[3] + U[1]) / 2;
    Real interp8 = (V[0] + V[1]) / 2;
    Real interp9 = (U[5] + U[2]) / 2;
    Real interp10 = (V[2] + V[1]) / 2;
    Real interp11 = interp7 * 2;
    Real interp12 = (V[1] - V[0]) / 2;
    Real interp13 = interp9 * 2;
    Real interp14 = (V[2] - V[1]) / 2;

    // dU^2/dx
    Real result_fd = (interp1 * interp1 - interp2 * interp2) * inv_dy;
    Real result_dc = gamma * (abs(interp3) / 2 * interp4 - abs(interp5) * interp6) * inv_dy;
    result += result_fd + result_dc;

    // dUV/dy
    result_fd = (interp7 * interp8 - interp9 * interp10) * inv_dx;
    result_dc = gamma * (abs(interp11) / 2 * interp12 - abs(interp13) / 2 * interp14) * inv_dx;
    result += result_fd + result_dc;
    return result;
}

__device__ Real laplacian(Real ar[6], Real inv_dx, Real inv_dy) {
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real result = (ar[0] - 2. * ar[1] + ar[2]) * inv_dx2 + (ar[3] - 2. * ar[1] + ar[4]) * inv_dy2;
    return result;
}

__device__ Real laplacian_5(Real ar[5], Real inv_dx, Real inv_dy) {
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real result = (ar[0] - 2. * ar[1] + ar[2]) * inv_dx2 + (ar[3] - 2. * ar[1] + ar[4]) * inv_dy2;
    return result;
}

__device__ Real sor_helper(Real ar[4], Real inv_dx, Real inv_dy) {
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real result = (ar[0] + ar[1]) * inv_dx2 + (ar[2] + ar[3]) * inv_dy2;
    return result;
}

__device__ Real convecton_uKEPS(Real U[2], Real T[5], Real inv_dx) {
    constexpr int METHOD = 0;

    if (METHOD == 0) {
        return convection_uT(U, T, inv_dx, 1);
    } else {
        // TODO
    }
}

__device__ Real convecton_vKEPS(Real V[2], Real T[5], Real inv_dy) {
    constexpr int METHOD = 0;
    if (METHOD == 0) {
        return convection_vT(V, T, inv_dy, 1);
    } else {
        // TODO
    }
}

__device__ Real laplacian_nu(Real ar[5], Real nu_i[2], Real nu_j[2], Real inv_dx, Real inv_dy, Real nu, Real coeff) {
    // nu_ij[0] -> nu_ij(i,j)
    // nu_ij[1] -> nu_i(i-1,j) or nu_j(i, j-1)
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real i_diff = (nu + nu_i[0]) * (ar[0] - ar[1]) - (nu + nu_i[1]) * (ar[1] - ar[2]);
    Real j_diff = (nu + nu_j[0]) * (ar[3] - ar[1]) - (nu + nu_j[1]) * (ar[1] - ar[4]);
    Real result = 1 / coeff * (i_diff * inv_dx2 + j_diff * inv_dy2);
    return result;
}

__device__ Real mean_strain_rate_squared(Real U[6], Real V[6], Real inv_dx, Real inv_dy) { 
    // U offsets:
    // 0,0 / -1,0 / 0,1 / -1,1 / 0,-1 / -1,-1
    // V offsets:
    // 0,0 / 0,-1 / 1,0 / 1, -1 / -1,0 / -1,-1
    constexpr int METHOD = 0;
    Real result = 0;
    if (METHOD == 0) {
        Real invdx2 = inv_dx * inv_dx;
        Real invdy2 = inv_dy * inv_dy;
        Real u_diff = U[0] - U[1];
        Real v_diff = V[0] - V[1];
        auto shear_1 = (U[2] + U[3] - U[4] - U[5]) * (0.25 * inv_dy);
        auto shear_2 = (V[2] + V[3] - V[4] - V[5]) * (0.25 * inv_dx);
        auto shear = shear_1 + shear_2;
        result = (u_diff * u_diff) * invdx2 + v_diff * v_diff * invdy2 + shear * shear;
    } else {
        // TODO 
    }
    return result;
}