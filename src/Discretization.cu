#include "hip/hip_runtime.h"
#pragma once
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "Utilities.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#define at(AR, I, J) AR[imax * (J) + (I)]

__device__ Real convection_u(Real U[6], Real V[6], Real inv_dx, Real inv_dy, Real gamma) {
    Real result = 0.0;
    Real interp1 = (U[0] + U[1]) / 2;
    Real interp2 = (U[2] + U[1]) / 2;
    Real interp3 = interp1 * 2;
    Real interp4 = (U[1] - U[0]) / 2;
    Real interp5 = interp2 * 2;
    Real interp6 = (U[2] - U[1]) / 2;

    Real interp7 = (V[0] + V[1]) / 2;
    Real interp8 = (U[3] + U[1]) / 2;
    Real interp9 = (V[5] + V[4]) / 2;
    Real interp10 = (U[4] + U[1]) / 2;
    Real interp11 = interp7 * 2;
    Real interp12 = (U[1] - U[3]) / 2;
    Real interp13 = interp9 * 2;
    Real interp14 = (U[4] - U[1]) / 2;

    // dU^2/dx
    Real result_fd = (interp1 * interp1 - interp2 * interp2) * inv_dx;
    Real result_dc = gamma * (abs(interp3) / 2 * interp4 - abs(interp5) / 2 * interp6) * inv_dx;
    result += result_fd + result_dc;

    // dUV/dy
    result_fd = (interp7 * interp8 - interp9 * interp10) * inv_dy;
    result_dc = gamma * (abs(interp11) / 2 * interp12 - abs(interp13) / 2 * interp14) * inv_dy;
    result += result_fd + result_dc;

    return result;
}

__device__  Real convection_uT(Real U[2], Real T[5], Real inv_dx, Real gamma) {
    Real result = 0.0;

    Real interp1 = (T[1] + T[0]) / 2;
    Real interp2 = (T[2] + T[1]) / 2;
    Real interp3 = (T[1] - T[0]) / 2;
    Real interp4 = (T[2] - T[1]) / 2;

    return inv_dx * ((U[1] * interp1 - U[0] * interp2) + gamma * (abs(U[1]) * interp3 - abs(U[0]) * interp4));
}

__device__  Real convection_vT(Real V[2], Real T[5], Real inv_dy, Real gamma) {
    Real result = 0.0;

    Real interp1 = (T[1] + T[3]) / 2;
    Real interp2 = (T[4] + T[1]) / 2;
    Real interp3 = (T[1] - T[3]) / 2;
    Real interp4 = (T[4] - T[1]) / 2;

    return inv_dy * ((V[1] * interp1 - V[0] * interp2) + gamma * (abs(V[1]) * interp3 - abs(V[0]) * interp4));
}

__device__  Real convection_v(Real U[6], Real V[6], Real inv_dx, Real inv_dy, Real gamma) {
    Real result = 0.0;
    Real interp1 = (V[3] + V[1]) / 2;
    Real interp2 = (V[4] + V[1]) / 2;
    Real interp3 = interp1 * 2;
    Real interp4 = (V[1] - V[3]) / 2;
    Real interp5 = interp2 * 2;
    Real interp6 = (V[4] - V[1]) / 2;

    Real interp7 = (U[3] + U[1]) / 2;
    Real interp8 = (V[0] + V[1]) / 2;
    Real interp9 = (U[5] + U[2]) / 2;
    Real interp10 = (V[2] + V[1]) / 2;
    Real interp11 = interp7 * 2;
    Real interp12 = (V[1] - V[0]) / 2;
    Real interp13 = interp9 * 2;
    Real interp14 = (V[2] - V[1]) / 2;

    // dU^2/dx
    Real result_fd = (interp1 * interp1 - interp2 * interp2) * inv_dy;
    Real result_dc = gamma * (abs(interp3) / 2 * interp4 - abs(interp5) * interp6) * inv_dy;
    result += result_fd + result_dc;

    // dUV/dy
    result_fd = (interp7 * interp8 - interp9 * interp10) * inv_dx;
    result_dc = gamma * (abs(interp11) / 2 * interp12 - abs(interp13) / 2 * interp14) * inv_dx;
    result += result_fd + result_dc;
    return result;
}

__device__  Real laplacian(Real ar[6], Real inv_dx, Real inv_dy) {
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real result = (ar[0] - 2. * ar[1] + ar[2]) * inv_dx2 + (ar[3] - 2. * ar[1] + ar[4]) * inv_dy2;
    return result;
}

__device__  Real laplacian_5(Real ar[5], Real inv_dx, Real inv_dy) {
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real result = (ar[0] - 2. * ar[1] + ar[2]) * inv_dx2 + (ar[3] - 2. * ar[1] + ar[4]) * inv_dy2;
    return result;
}