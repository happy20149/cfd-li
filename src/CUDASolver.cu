#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "CUDASolver.hpp"
#include "Discretization.cu"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLK_SIZE 128
#define BLK_SIZE_2D 32

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#define chk(ans)                                                                                                       \
    { gpuAssert((ans), __FILE__, __LINE__); }

int get_num_blks(int size) { return (size + BLK_SIZE - 1) / BLK_SIZE; }
dim3 get_num_blks_2d(int size_x, int size_y) {
    return dim3((size_x + BLK_SIZE_2D - 1) / BLK_SIZE_2D, (size_y + BLK_SIZE_2D - 1) / BLK_SIZE_2D);
}

template <typename T> inline void malloc_assign(T *dev_ptr, T val) {
    chk(hipMalloc(&dev_ptr, sizeof(T)));
    chk(hipMemcpy(dev_ptr, &val, 1, hipMemcpyHostToDevice));
}

__global__ void saxpy(Real *a, Real *x, Real *y, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {

        y[i] = *a * x[i] + y[i];
    }
}

__global__ void smaxpy(Real *a, Real *x, Real *y, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        y[i] = -(*a) * x[i] + y[i];
    }
}

__global__ void saxpy2(Real *a, Real *x, Real *y, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        y[i] = x[i] + *a * y[i];
    }
}

__global__ void vec_dot_vec(Real *a, Real *b, Real *o, int size) {
    __shared__ Real sdata[BLK_SIZE];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        *o = 0;
    }
    if (i < size) {
        sdata[tid] = a[i] * b[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(o, sdata[0]);
    }
}
// https://www.nvidia.com/docs/IO/66889/nvr-2008-004.pdf
__global__ void spmv_dia(Real *data, int *offsets, int num_rows, int num_cols, int num_diags, Real *x, Real *y) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0;
        y[row] = 0;
        for (int n = 0; n < num_diags; n++) {
            int col = row + offsets[n];
            float val = data[num_rows * n + row];
            if (col >= 0 && col < num_cols) dot += val * x[col];
        }
        y[row] += dot;
    }
}

__global__ void scalar_div(Real *num, Real *denom, Real *o) { *o = *num / *denom; }
__global__ void scalar_cpy(Real *dst, Real *src) { *dst = *src; }

void solve_pcg(Real *A, int *A_offsets, int num_diag, Real *x, Real *b, Real *q, Real *d, Real *r, Real *r_dot_r_old,
               Real *r_dot_r, Real *z, Real *cg_beta, Real &delta_new, Real *cg_alpha, Real *d_dot_q, int precondition,
               Real *M, int *M_offsets, int m_num_diag, uint32_t &it, uint32_t max_iter, Real eps, int vec_size) {
    int num_blocks = get_num_blks(vec_size);
    hipMemcpy(r, b, vec_size * sizeof(Real), hipMemcpyDeviceToDevice);
    hipMemset(x, 0, vec_size * sizeof(Real));
    if (precondition != -1) {
        spmv_dia<<<num_blocks, BLK_SIZE>>>(M, M_offsets, vec_size, vec_size, m_num_diag, r, d);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, d, r_dot_r, vec_size);
    } else {
        hipMemcpy(d, b, vec_size * sizeof(Real), hipMemcpyDeviceToDevice);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, r, r_dot_r, vec_size);
    }
    hipMemcpy(&delta_new, r_dot_r, sizeof(Real), hipMemcpyDeviceToHost);
    Real cond = delta_new * eps * eps;
    it = 0;
    while (it < max_iter && delta_new > cond) {
        // q <- A * d
        spmv_dia<<<num_blocks, BLK_SIZE>>>(A, A_offsets, vec_size, vec_size, num_diag, d, q);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(d, q, d_dot_q, vec_size);
        // cg_alpha <- r_dot_r / d_dot_q
        scalar_div<<<1, 1>>>(r_dot_r, d_dot_q, cg_alpha);
        // x <- x + cg_alpha * d
        saxpy<<<num_blocks, BLK_SIZE>>>(cg_alpha, d, x, vec_size);
        // r <- r - cg_alpha * q
        smaxpy<<<num_blocks, BLK_SIZE>>>(cg_alpha, q, r, vec_size);
        scalar_cpy<<<1, 1>>>(r_dot_r_old, r_dot_r);
        if (precondition != -1) {
            // z <- M * r
            spmv_dia<<<num_blocks, BLK_SIZE>>>(M, M_offsets, vec_size, vec_size, m_num_diag, r, z);
            vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, z, r_dot_r, vec_size);
        } else {
            vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, r, r_dot_r, vec_size);
        }
        // cg_beta <- r_dot_r / r_dot_r_old
        scalar_div<<<1, 1>>>(r_dot_r, r_dot_r_old, cg_beta);
        if (precondition != -1) {
            // d <- z + cg_beta * d
            saxpy2<<<num_blocks, BLK_SIZE>>>(cg_beta, z, d, vec_size);
        } else {
            // d <- r + cg_beta *d
            saxpy2<<<num_blocks, BLK_SIZE>>>(cg_beta, r, d, vec_size);
        }
        it++;
        hipMemcpy(&delta_new, r_dot_r, sizeof(Real), hipMemcpyDeviceToHost);
    }
}

__global__ void sor_iter(Real *P, Real *RS, Real coeff, int *cell_type, int imax, int jmax, Real omega, Real inv_dx,
                         Real inv_dy, int parity) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }

    if (parity == 0 && ((i + j) % 2) == 0) {
        return;
    } else if (parity == 1 && ((i + j) % 2) == 1) {
        return;
    }
    Real p_stencil[4] = {at(P, i + 1, j), at(P, i - 1, j), at(P, i, j + 1), at(P, i, j - 1)};
    at(P, i, j) = (1 - omega) * at(P, i, j) + coeff * (sor_helper(p_stencil, inv_dx, inv_dy) - at(RS, i, j));
}

__global__ void calc_residual(Real *P, Real *RS, int *cell_type, int imax, int jmax, Real inv_dx, Real inv_dy,
                              Real *residual) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }

    Real rloc = 0;
    Real p_laplacian[5] = {at(P, i + 1, j), at(P, i, j), at(P, i - 1, j), at(P, i, j + 1), at(P, i, j - 1)};

    Real val = laplacian_5(p_laplacian, inv_dx, inv_dy) - at(RS, i, j);
    rloc += val * val;
    at(residual, i, j) = rloc;
}

__global__ void reduce_residual(Real *residual, Real *o, int size) {
    __shared__ Real sdata[BLK_SIZE];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        *o = 0;
    }
    if (i < size) {
        sdata[tid] = residual[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(o, sdata[0]);
    }
}

__global__ void negate_p(Real *p, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        p[i] = -p[i];
    }
}

__device__ Real vel_kernel(Real fg, Real dt, Real p[2], Real inv_dxy) { return fg - dt * inv_dxy * (p[1] - p[0]); }

__global__ void calc_vel(Real *u, Real *v, Real *p, Real *f, Real *g, int *cell_type, Real dt, int imax, int jmax,
                         Real dx, Real dy) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;

    float p_diff_u[2] = {at(p, i, j), at(p, i + 1, j)};
    float p_diff_v[2] = {at(p, i, j), at(p, i, j + 1)};
    /*  at(u, i, j) = vel_kernel(at(f, i, j), *dt, p_diff_u, inv_dx);
      at(v, i, j) = vel_kernel(at(g, i, j), *dt, p_diff_v, inv_dy);*/
    at(u, i, j) = at(f, i, j) - dt * inv_dx * (p_diff_u[1] - p_diff_u[0]);
    at(v, i, j) = at(g, i, j) - dt * inv_dy * (p_diff_v[1] - p_diff_v[0]);
}

__global__ void enforce_boundary(Real *u, int *row_start, int *col_idx, Real *mat, Real *rhs_vec, int size) {
    uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < size) {
        Real sum = 0;
        for (int j = row_start[row]; j < row_start[row + 1]; j++) {
            sum += mat[j] * u[col_idx[j]];
        }
        u[row] = sum + 2 * rhs_vec[row];
    }
}

void uv_boundary(Real *u, Real *v, int *row_start_u, int *row_start_v, int *col_idx_u, int *col_idx_v, Real *mat_u,
                 Real *mat_v, Real *rhs_vec_u, Real *rhs_vec_v, int size) {
    int num_blks(get_num_blks(size));
    enforce_boundary<<<num_blks, BLK_SIZE>>>(u, row_start_u, col_idx_u, mat_u, rhs_vec_u, size);
    enforce_boundary<<<num_blks, BLK_SIZE>>>(v, row_start_v, col_idx_v, mat_v, rhs_vec_v, size);
}

// See https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
__global__ void reduce_abs_max(Real *input, Real *output, int size) {
    extern __shared__ Real sdata[];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    Real curr_max = (i < size) ? fabsf(input[i]) : 0;
    if (i + blockDim.x < size) {
        curr_max = fmaxf(curr_max, fabsf(input[i + blockDim.x]));
    }
    sdata[tid] = curr_max;
    __syncthreads();
    for (uint32_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

void t_boundary(Real *t, int *row_start_t, int *col_idx_t, Real *mat_t, Real *rhs_vec_t, int size) {
    int num_blks(get_num_blks(size));
    enforce_boundary<<<num_blks, BLK_SIZE>>>(t, row_start_t, col_idx_t, mat_t, rhs_vec_t, size);
}

__global__ void calc_t(Real *u, Real *v, Real dx, Real dy, Real *t_new, Real *t_old, int *cell_type, Real alpha,
                       Real dt, Real gamma, int imax, int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real u_stencil[2] = {at(u, i - 1, j), at(u, i, j)};
    Real v_stencil[2] = {at(v, i, j - 1), at(v, i, j)};

    Real t_laplacian[5] = {at(t_old, i + 1, j), at(t_old, i, j), at(t_old, i - 1, j), at(t_old, i, j + 1),
                           at(t_old, i, j - 1)};

    at(t_new, i, j) = at(t_old, i, j) + dt * (alpha * laplacian_5(t_laplacian, inv_dx, inv_dy) -
                                              convection_uT(u_stencil, t_laplacian, inv_dx, gamma) -
                                              convection_vT(v_stencil, t_laplacian, inv_dy, gamma));
}

__global__ void calc_fg(Real *f, Real *g, Real *u, Real *v, bool calc_temp, Real dx, Real dy, Real *t, int *cell_type,
                        Real dt, Real gamma, Real nu, Real beta, Real gx, Real gy, int imax, int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    // 5-point + 1 stencil for U and V
    Real u_stencil[6] = {at(u, i + 1, j), at(u, i, j),     at(u, i - 1, j),
                         at(u, i, j + 1), at(u, i, j - 1), at(u, i - 1, j + 1)};
    Real v_stencil[6] = {at(v, i + 1, j), at(v, i, j),     at(v, i - 1, j),
                         at(v, i, j + 1), at(v, i, j - 1), at(v, i + 1, j - 1)};

    // Calculate fluxes
    at(f, i, j) = at(u, i, j) + dt * (nu * laplacian(u_stencil, inv_dx, inv_dy) -
                                      convection_u(u_stencil, v_stencil, inv_dx, inv_dy, gamma));
    at(g, i, j) = at(v, i, j) + dt * (nu * laplacian(v_stencil, inv_dx, inv_dy) -
                                      convection_v(u_stencil, v_stencil, inv_dx, inv_dy, gamma));

    if (calc_temp) {
        Real term1 = at(t, i, j) + at(t, i + 1, j);
        Real term2 = at(t, i, j) + at(t, i, j + 1);
        at(f, i, j) -= beta * dt / 2 * (term1)*gx;
        at(g, i, j) -= beta * dt / 2 * (term2)*gy;
    }
}

__global__ void fg_boundary(Real *f, Real *g, Real *u, Real *v, int imax, int jmax, uint32_t *neighborhood,
                            int *cell_type) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 1) {
        return;
    }
    uint32_t type = at(neighborhood, i, j) >> 8;
    uint32_t neighbors = at(neighborhood, i, j) & 0xFF;

    if ((neighbors & 0x1) == 1) {
        at(f, i, j) = at(u, i, j);
    }
    if ((neighbors & 0x2) == 2) {
        at(f, i - 1, j) = at(u, i - 1, j);
    }
    if ((neighbors & 0x4) == 4) {
        at(g, i, j) = at(v, i, j);
    }
    if ((neighbors & 0x8) == 8) {
        at(g, i, j - 1) = at(v, i, j - 1);
    }
}

__global__ void p_boundary(Real *p, int imax, int jmax, uint32_t *neighborhood, int *cell_type, Real PI) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 1) {
        return;
    }
    uint32_t type = at(neighborhood, i, j) >> 8;
    uint32_t neighbors = at(neighborhood, i, j) & 0xFF;
    if (type == 0) { // Outlet
        at(p, i, j) = PI;
    } else {
        int diag = 0;
        if ((neighbors & 0x10) == 16) { // Right + top
            diag = 1;
            at(p, i, j) = (at(p, i + 1, j) + at(p, i, j + 1)) / 2;
        }
        if ((neighbors & 0x20) == 32) { // Right + bottom
            diag = 1;
            at(p, i, j) = (at(p, i + 1, j) + at(p, i, j - 1)) / 2;
        }
        if ((neighbors & 0x40) == 64) { // Left + top
            diag = 1;
            at(p, i, j) = (at(p, i - 1, j) + at(p, i, j + 1)) / 2;
        }
        if ((neighbors & 0x80) == 128) { // Left + bottom
            diag = 1;
            at(p, i, j) = (at(p, i - 1, j) + at(p, i, j - 1)) / 2;
        }
        if (!diag) {
            if ((neighbors & 0x1) == 1) { // Right
                at(p, i, j) = at(p, i + 1, j);
            }
            if ((neighbors & 0x2) == 2) { // Left
                at(p, i, j) = at(p, i - 1, j);
            }
            if ((neighbors & 0x4) == 4) { // Top
                at(p, i, j) = at(p, i, j + 1);
            }
            if ((neighbors & 0x8) == 8) { // Bottom
                at(p, i, j) = at(p, i, j - 1);
            }
        }
    }
}

void solve_sor(Real *P, Real *P_tmp, Real *P_residual, Real *P_residual_out, uint32_t *neighborhood, int imax, int jmax,
               Real *RS, int *cell_type, uint32_t &it, uint32_t max_iter, Real dx, Real dy, Real PI, Real tolerance,
               Real &res, int num_fluid_cells) {
    it = 0;
    const Real omega = 1.7;
    auto grid_size = imax * jmax;
    Real coeff = omega / (2 * (1 / (dx * dx) + 1 / (dy * dy)));
    dim3 blk_size_2d(BLK_SIZE_2D, BLK_SIZE_2D);
    dim3 num_blks_2d = get_num_blks_2d(imax, jmax);
    int num_blks_1d(get_num_blks(grid_size));
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    res = REAL_MAX;
    while (it < max_iter && res > tolerance) {
        sor_iter<<<num_blks_2d, blk_size_2d>>>(P, RS, coeff, cell_type, imax, jmax, omega, inv_dx, inv_dy, 0);
        sor_iter<<<num_blks_2d, blk_size_2d>>>(P, RS, coeff, cell_type, imax, jmax, omega, inv_dx, inv_dy, 1);
        /* std::vector<Real> Pcpu2(grid_size);
         hipMemcpy(Pcpu2.data(), P, grid_size * sizeof(Real), hipMemcpyDeviceToHost);*/

        hipMemset(P_residual, 0, grid_size * sizeof(Real));
        calc_residual<<<num_blks_2d, blk_size_2d>>>(P, RS, cell_type, imax, jmax, inv_dx, inv_dy, P_residual);
        reduce_residual<<<num_blks_1d, BLK_SIZE>>>(P_residual, P_residual_out, grid_size);
        p_boundary<<<num_blks_2d, blk_size_2d>>>(P, imax, jmax, neighborhood, cell_type, PI);
        hipMemcpy(&res, P_residual_out, sizeof(Real), hipMemcpyDeviceToHost);
        /*   std::vector<Real> Pcpu1(grid_size);
           hipMemcpy(Pcpu1.data(), P, grid_size * sizeof(Real), hipMemcpyDeviceToHost);*/
        res = std::sqrt(res / num_fluid_cells);
        it++;
    }
}

__global__ void calc_rs(Real *f, Real *g, Real *rs, Real dx, Real dy, int imax, int jmax, Real dt, int *cell_type) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);

    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }

    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    float f_diff[2] = {at(f, i, j), at(f, i - 1, j)};
    float g_diff[2] = {at(g, i, j), at(g, i, j - 1)};

    float df = inv_dx * (f_diff[0] - f_diff[1]);
    float dg = inv_dy * (g_diff[0] - g_diff[1]);
    at(rs, i, j) = (df + dg) * 1 / dt;
}

Real calculate_dt(int imax, int jmax, Real *u, Real *v, Real *u_residual, Real *v_residual, Real dx, Real dy, Real tau,
                  Real nu, Real alpha, bool calc_temp) {
    // Calculate uv max
    int size = imax * jmax;
    int num_blks(get_num_blks(size));
    Real u_max_abs = 0;
    Real v_max_abs = 0;

    Real dx2 = dx * dx;
    Real dy2 = dy * dy;
    int smemsize = min(BLK_SIZE, size);
    std::vector<Real> ucpu(imax * jmax);
    std::vector<Real> vcpu(imax * jmax);
    std::vector<Real> ures(imax * jmax);
    std::vector<Real> vres(imax * jmax);

    reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(u, u_residual, size);
    reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(v, v_residual, size);
    while (num_blks != 1) {
        size = ceil(size / float(BLK_SIZE));
        smemsize = min(BLK_SIZE, size);
        num_blks = get_num_blks(size);
        reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(u_residual, u_residual, size);
        reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(v_residual, v_residual, size);
    }
    hipMemcpy(&u_max_abs, u_residual, sizeof(Real), hipMemcpyDeviceToHost);
    hipMemcpy(&v_max_abs, v_residual, sizeof(Real), hipMemcpyDeviceToHost);
    Real min_cond = std::min(dx / u_max_abs, dy / v_max_abs);
    if (nu != 0) {
        Real cond_spatial = 1.0 / (2.0 * nu) * ((dx2 * dy2) / (dx2 + dy2));
        min_cond = std::min(min_cond, cond_spatial);
    }
    if (calc_temp) {
        Real inv_dx = 1 / dx;
        Real inv_dx2 = inv_dx * inv_dx;
        Real inv_dy = 1 / dy;
        Real inv_dy2 = inv_dy * inv_dy;
        float cond_temp = 1 / (2 * alpha * (inv_dx2 + inv_dy2));
        min_cond = std::min(min_cond, cond_temp);
    }
    return tau * min_cond;
}
void CudaSolver::initialize() {
    auto grid_x = _grid.imaxb();
    auto grid_y = _grid.jmaxb();
    auto grid_size = grid_x * grid_y;
    build_pcg_matrix(_field, _grid, _boundaries, A_pcg, U_pcg, V_pcg, T_pcg, U_RHS, V_RHS, T_RHS, U_fixed, V_fixed,
                     T_fixed);
    // Preprocess
    std::vector<int> is_fluid(_grid.imaxb() * _grid.jmaxb(), 0);
    for (const auto &current_cell : _grid.fluid_cells()) {
        int i = current_cell->i();
        int j = current_cell->j();
        is_fluid[_grid.imaxb() * j + i] = 1;
    }

    std::vector<BoundaryData> neighbors(_grid.imaxb() * _grid.jmaxb());
    for (const auto &boundary : _boundaries) {
        uint32_t type = boundary->get_type();
        auto cells = boundary->_cells;
        for (auto &cell : *cells) {
            int i = cell->i();
            int j = cell->j();
            BoundaryData data;
            uint32_t type = boundary->get_type();
            data.neighborhood |= type << 8;
            // data.idx = _grid.imaxb() * j + i;
            if (cell->is_border(border_position::RIGHT)) {
                data.neighborhood |= 1;
            }
            if (cell->is_border(border_position::LEFT)) {
                data.neighborhood |= 2;
            }
            if (cell->is_border(border_position::TOP)) {
                data.neighborhood |= 4;
            }
            if (cell->is_border(border_position::BOTTOM)) {
                data.neighborhood |= 8;
            }
            if (cell->is_border(border_position::RIGHT) && cell->is_border(border_position::TOP)) {
                data.neighborhood |= 16;
            }
            if (cell->is_border(border_position::RIGHT) && cell->is_border(border_position::BOTTOM)) {
                data.neighborhood |= 32;
            }
            if (cell->is_border(border_position::LEFT) && cell->is_border(border_position::TOP)) {
                data.neighborhood |= 64;
            }
            if (cell->is_border(border_position::LEFT) && cell->is_border(border_position::BOTTOM)) {
                data.neighborhood |= 128;
            }
            neighbors[j * _grid.imaxb() + i] = data;
        }
    }
    DiagonalSparseMatrix<Real> A_matrix_diag =
        create_diagonal_matrix(A_pcg, _grid.imaxb(), _grid.jmaxb(), {-_grid.imaxb(), -1, 0, 1, _grid.imaxb()});
    DiagonalSparseMatrix<Real> A_precond_diag;
    if (_preconditioner != -1) {
        A_precond_diag = create_preconditioner_spai(A_pcg, _grid, _preconditioner);
    }
    num_offsets_a = A_matrix_diag.offsets.size();
    num_offsets_m = A_precond_diag.offsets.size();
    auto t_matrix_data = T_fixed.value.data();
    auto t_matrix_size = T_fixed.value.size();
    auto t_row_start_data = T_fixed.rowstart.data();
    auto t_row_start_size = T_fixed.rowstart.size();
    auto t_col_idx_data = T_fixed.colindex.data();
    auto t_col_idx_size = T_fixed.colindex.size();
    auto t_rhs_data = T_RHS.data();
    auto t_rhs_size = T_RHS.size();
    auto u_matrix_data = U_fixed.value.data();
    auto u_matrix_size = U_fixed.value.size();
    auto v_matrix_data = V_fixed.value.data();
    auto v_matrix_size = V_fixed.value.size();
    auto u_row_start_data = U_fixed.rowstart.data();
    auto u_row_start_size = U_fixed.rowstart.size();
    auto u_col_idx_data = U_fixed.colindex.data();
    auto u_col_idx_size = U_fixed.colindex.size();
    auto v_row_start_data = V_fixed.rowstart.data();
    auto v_row_start_size = V_fixed.rowstart.size();
    auto v_col_idx_data = V_fixed.colindex.data();
    auto v_col_idx_size = V_fixed.colindex.size();
    auto u_rhs_data = U_RHS.data();
    auto u_rhs_size = U_RHS.size();
    auto v_rhs_data = V_RHS.data();
    auto v_rhs_size = V_RHS.size();

    hipMalloc(&U, grid_size * sizeof(Real));
    hipMalloc(&V, grid_size * sizeof(Real));
    hipMalloc(&F, grid_size * sizeof(Real));
    hipMalloc(&G, grid_size * sizeof(Real));
    hipMalloc(&P, grid_size * sizeof(Real));
    hipMalloc(&P_temp, grid_size * sizeof(Real));

    hipMalloc(&RS, grid_size * sizeof(Real));
    hipMalloc(&U_residual, grid_size * sizeof(Real));
    hipMalloc(&V_residual, grid_size * sizeof(Real));
    hipMalloc(&P_residual, grid_size * sizeof(Real));
    hipMalloc(&cell_type, grid_size * sizeof(int));
    hipMalloc(&row_start_u, u_row_start_size * sizeof(int));
    hipMalloc(&row_start_v, v_row_start_size * sizeof(int));
    hipMalloc(&row_start_t, t_row_start_size * sizeof(int));
    hipMalloc(&col_idx_u, u_col_idx_size * sizeof(int));
    hipMalloc(&col_idx_v, v_col_idx_size * sizeof(int));
    hipMalloc(&mat_u, u_matrix_size * sizeof(Real));
    hipMalloc(&mat_v, v_matrix_size * sizeof(Real));
    hipMalloc(&rhs_vec_u, u_rhs_size * sizeof(Real));
    hipMalloc(&rhs_vec_v, v_rhs_size * sizeof(Real));
    hipMalloc(&neighborhood, neighbors.size() * sizeof(uint32_t));

    hipMalloc(&A, A_matrix_diag.data.size() * sizeof(Real));
    hipMalloc(&A_offsets, A_matrix_diag.offsets.size() * sizeof(uint32_t));
    if (_preconditioner != -1) {
        hipMalloc(&M, A_precond_diag.data.size() * sizeof(Real));
        hipMalloc(&M_offsets, A_precond_diag.offsets.size() * sizeof(uint32_t));
    }
    hipMalloc(&q, grid_size * sizeof(Real));
    hipMalloc(&d, grid_size * sizeof(Real));
    hipMalloc(&r, grid_size * sizeof(Real));
    hipMalloc(&z, grid_size * sizeof(Real));
    hipMalloc(&r_dot_r, sizeof(Real));
    hipMalloc(&r_dot_r_old, sizeof(Real));
    hipMalloc(&d_dot_q, sizeof(Real));
    hipMalloc(&p_residual_out, sizeof(Real));
    hipMalloc(&cg_alpha, sizeof(Real));
    hipMalloc(&cg_beta, sizeof(Real));

    chk(hipMemcpy(U, _field._U._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(V, _field._V._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(P, _field._P._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(mat_u, u_matrix_data, u_matrix_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(row_start_u, u_row_start_data, u_row_start_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(col_idx_u, u_col_idx_data, u_col_idx_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(rhs_vec_u, u_rhs_data, u_rhs_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(mat_v, v_matrix_data, v_matrix_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(row_start_v, v_row_start_data, v_row_start_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(col_idx_v, v_col_idx_data, v_col_idx_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(rhs_vec_v, v_rhs_data, v_rhs_size * sizeof(Real), hipMemcpyHostToDevice));

    chk(hipMemcpy(neighborhood, neighbors.data(), neighbors.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

    chk(hipMemcpy(A, A_matrix_diag.data.data(), A_matrix_diag.data.size() * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(A_offsets, A_matrix_diag.offsets.data(), A_matrix_diag.offsets.size() * sizeof(int),
                   hipMemcpyHostToDevice));
    if (_calc_temp) {
        hipMalloc(&T, grid_size * sizeof(Real));
        hipMalloc(&T_temp, grid_size * sizeof(Real));
        hipMalloc(&mat_t, t_matrix_size * sizeof(Real));
        hipMalloc(&rhs_vec_t, t_rhs_size * sizeof(Real));
        hipMalloc(&col_idx_t, t_col_idx_size * sizeof(int));

        hipMemcpy(mat_t, t_matrix_data, t_matrix_size * sizeof(Real), hipMemcpyHostToDevice);
        hipMemcpy(row_start_t, t_row_start_data, t_row_start_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(col_idx_t, t_col_idx_data, t_col_idx_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(rhs_vec_t, t_rhs_data, t_rhs_size * sizeof(Real), hipMemcpyHostToDevice);
        chk(hipMemcpy(T, _field._T._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    }

    if (_preconditioner != -1) {
        chk(hipMemcpy(M, A_precond_diag.data.data(), A_precond_diag.data.size() * sizeof(Real),
                       hipMemcpyHostToDevice));
        chk(hipMemcpy(M_offsets, A_precond_diag.offsets.data(), A_precond_diag.offsets.size() * sizeof(int),
                       hipMemcpyHostToDevice));
    }
    chk(hipMemcpy(cell_type, is_fluid.data(), is_fluid.size() * sizeof(int), hipMemcpyHostToDevice));
}
void CudaSolver::solve_pre_pressure(Real &dt) {
    auto grid_x = _grid.imaxb();
    auto grid_y = _grid.jmaxb();
    auto grid_size = grid_x * grid_y;
    dim3 num_blks_1d(get_num_blks(grid_size));
    dim3 blk_size_2d(BLK_SIZE_2D, BLK_SIZE_2D);
    dim3 num_blks_2d = get_num_blks_2d(grid_x, grid_y);
    dt = calculate_dt(_grid.imaxb(), _grid.jmaxb(), U, V, U_residual, V_residual, _grid.dx(), _grid.dy(), _field._tau,
                      _field._nu, _field._alpha, _calc_temp);
    _field._dt = dt;

    uv_boundary(U, V, row_start_u, row_start_v, col_idx_u, col_idx_v, mat_u, mat_v, rhs_vec_u, rhs_vec_v, grid_size);
    if (_calc_temp) {
        t_boundary(T, row_start_t, col_idx_t, mat_t, rhs_vec_t, grid_size);
        chk(hipMemcpy(T_temp, T, grid_size * sizeof(Real), hipMemcpyDeviceToDevice));
        calc_t<<<num_blks_2d, blk_size_2d>>>(U, V, _grid.dx(), _grid.dy(), T, T_temp, cell_type, _field._alpha, dt,
                                             _discretization._gamma, _grid.imaxb(), _grid.jmaxb());
    }
    calc_fg<<<num_blks_2d, blk_size_2d>>>(F, G, U, V, _calc_temp, _grid.dx(), _grid.dy(), T, cell_type, dt,
                                          _discretization._gamma, _field._nu, _field._beta, _field._gx, _field._gy,
                                          grid_x, grid_y);
    fg_boundary<<<num_blks_2d, blk_size_2d>>>(F, G, U, V, grid_x, grid_y, neighborhood, cell_type);
    calc_rs<<<num_blks_2d, blk_size_2d>>>(F, G, RS, _grid.dx(), _grid.dy(), grid_x, grid_y, dt, cell_type);
}

void CudaSolver::solve_pressure(Real &res, uint32_t &it) {
    if (solver_type == SolverType::PCG) {
        auto grid_x = _grid.imaxb();
        auto grid_y = _grid.jmaxb();
        auto grid_size = grid_x * grid_y;
        int num_blks(get_num_blks(grid_size));
        solve_pcg(A, A_offsets, num_offsets_a, P, RS, q, d, r, r_dot_r_old, r_dot_r, z, cg_beta, res, cg_alpha, d_dot_q,
                  _preconditioner, M, M_offsets, num_offsets_m, it, _max_iter, _tolerance,
                  _grid.imaxb() * _grid.jmaxb());
        negate_p<<<num_blks, BLK_SIZE>>>(P, grid_size);
    } else if (solver_type == SolverType::SOR) {
        solve_sor(P, P_temp, P_residual, p_residual_out, neighborhood, _grid.imaxb(), _grid.jmaxb(), RS, cell_type, it,
                  _max_iter, _grid.dx(), _grid.dy(), _field._PI, _tolerance, res, _grid.fluid_cells().size()); 
    }
}

void CudaSolver::solve_post_pressure() {
    auto grid_x = _grid.imaxb();
    auto grid_y = _grid.jmaxb();
    auto grid_size = grid_x * grid_y;
    int num_blks(get_num_blks(grid_size));
    dim3 blk_size_2d(BLK_SIZE_2D, BLK_SIZE_2D);
    dim3 num_blks_2d = get_num_blks_2d(grid_x, grid_y);
    calc_vel<<<num_blks_2d, blk_size_2d>>>(U, V, P, F, G, cell_type, _field._dt, grid_x, grid_y, _grid.dx(),
                                           _grid.dy());
    chk(hipMemcpy(_field._U._container.data(), U, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    chk(hipMemcpy(_field._V._container.data(), V, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    chk(hipMemcpy(_field._P._container.data(), P, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    if (_calc_temp) {
        chk(hipMemcpy(_field._T._container.data(), T, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    }
}

CudaSolver::~CudaSolver() {
    hipFree(U);
    hipFree(V);
    hipFree(F);
    hipFree(G);
    hipFree(P);
    hipFree(T);
    hipFree(T_temp);
    hipFree(RS);
    hipFree(U_residual);
    hipFree(V_residual);
    hipFree(P_residual);
    hipFree(cell_type);
    hipFree(P);
    hipFree(P_temp);
    hipFree(row_start_u);
    hipFree(row_start_v);
    hipFree(row_start_t);
    hipFree(col_idx_u);
    hipFree(col_idx_v);
    hipFree(col_idx_t);
    hipFree(mat_u);
    hipFree(mat_v);
    hipFree(mat_t);
    hipFree(rhs_vec_u);
    hipFree(rhs_vec_v);
    hipFree(rhs_vec_t);
    hipFree(neighborhood);
    hipFree(A);
    hipFree(A_offsets);
    if (_preconditioner != -1) {
        hipFree(M);
        hipFree(M_offsets);
    }
    hipFree(q);
    hipFree(d);
    hipFree(r);
    hipFree(z);
    hipFree(r_dot_r);
    hipFree(r_dot_r_old);
    hipFree(d_dot_q);
    hipFree(p_residual_out);
    hipFree(cg_alpha);
    hipFree(cg_beta);
}
