#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "CUDASolver.hpp"
#include "Discretization.cu"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLK_SIZE 128
#define BLK_SIZE_2D 32

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#define chk(ans)                                                                                                       \
    { gpuAssert((ans), __FILE__, __LINE__); }

int get_num_blks(int size) { return (size + BLK_SIZE - 1) / BLK_SIZE; }
dim3 get_num_blks_2d(int size_x, int size_y) {
    return dim3((size_x + BLK_SIZE_2D - 1) / BLK_SIZE_2D, (size_y + BLK_SIZE_2D - 1) / BLK_SIZE_2D);
}

template <typename T> inline void malloc_assign(T *dev_ptr, T val) {
    chk(hipMalloc(&dev_ptr, sizeof(T)));
    chk(hipMemcpy(dev_ptr, &val, 1, hipMemcpyHostToDevice));
}

__global__ void init(Real *a, Real val, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        a[i] = val;
    }
}

__global__ void saxpy(Real *a, Real *x, Real *y, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {

        y[i] = *a * x[i] + y[i];
    }
}

__global__ void smaxpy(Real *a, Real *x, Real *y, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        y[i] = -(*a) * x[i] + y[i];
    }
}

__global__ void saxpy2(Real *a, Real *x, Real *y, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        y[i] = x[i] + *a * y[i];
    }
}

__global__ void vec_dot_vec(Real *a, Real *b, Real *o, int size) {
    __shared__ Real sdata[BLK_SIZE];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        *o = 0;
    }
    if (i < size) {
        sdata[tid] = a[i] * b[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(o, sdata[0]);
    }
}
// https://www.nvidia.com/docs/IO/66889/nvr-2008-004.pdf
__global__ void spmv_dia(Real *data, int *offsets, int num_rows, int num_cols, int num_diags, Real *x, Real *y) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows) {
        Real dot = 0;
        y[row] = 0;
        for (int n = 0; n < num_diags; n++) {
            int col = row + offsets[n];
            Real val = data[num_rows * n + row];
            if (col >= 0 && col < num_cols) dot += val * x[col];
        }
        y[row] += dot;
    }
}

// See https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
__global__ void reduce_abs_max(Real *input, Real *output, int size) {
    extern __shared__ Real sdata[];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    Real curr_max = (i < size) ? real_abs(input[i]) : 0;
    if (i + blockDim.x < size) {
        curr_max = real_max(curr_max, real_abs(input[i + blockDim.x]));
    }
    sdata[tid] = curr_max;
    __syncthreads();
    for (uint32_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = real_max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

__global__ void reduce_min(Real *input, Real *output, int size) {
    extern __shared__ Real sdata[];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    Real curr_max = (i < size) ? input[i] : 0;
    if (i + blockDim.x < size) {
        curr_max = real_min(curr_max, (input[i + blockDim.x]));
    }
    sdata[tid] = curr_max;
    __syncthreads();
    for (uint32_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = real_min(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

__global__ void reduce_max(Real *input, Real *output, int size) {
    extern __shared__ Real sdata[];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    Real curr_max = (i < size) ? input[i] : 0;
    if (i + blockDim.x < size) {
        curr_max = real_max(curr_max, (input[i + blockDim.x]));
    }
    sdata[tid] = curr_max;
    __syncthreads();
    for (uint32_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = real_max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

__global__ void scalar_div(Real *num, Real *denom, Real *o) { *o = *num / *denom; }
__global__ void scalar_cpy(Real *dst, Real *src) { *dst = *src; }

void solve_pcg(Real *A, int *A_offsets, int num_diag, Real *x, Real *b, Real *q, Real *d, Real *r, Real *r_dot_r_old,
               Real *r_dot_r, Real *z, Real *cg_beta, Real &delta_new, Real *cg_alpha, Real *d_dot_q, int precondition,
               Real *M, int *M_offsets, int m_num_diag, uint32_t &it, uint32_t max_iter, Real eps, int vec_size) {
    int num_blocks = get_num_blks(vec_size);
    hipMemcpy(r, b, vec_size * sizeof(Real), hipMemcpyDeviceToDevice);
    hipMemset(x, 0, vec_size * sizeof(Real));
    if (precondition != -1) {
        spmv_dia<<<num_blocks, BLK_SIZE>>>(M, M_offsets, vec_size, vec_size, m_num_diag, r, d);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, d, r_dot_r, vec_size);
    } else {
        hipMemcpy(d, b, vec_size * sizeof(Real), hipMemcpyDeviceToDevice);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, r, r_dot_r, vec_size);
    }
    hipMemcpy(&delta_new, r_dot_r, sizeof(Real), hipMemcpyDeviceToHost);
    Real cond = delta_new * eps * eps;
    it = 0;
    while (it < max_iter && delta_new > cond) {
        // q <- A * d
        spmv_dia<<<num_blocks, BLK_SIZE>>>(A, A_offsets, vec_size, vec_size, num_diag, d, q);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(d, q, d_dot_q, vec_size);
        // cg_alpha <- r_dot_r / d_dot_q
        scalar_div<<<1, 1>>>(r_dot_r, d_dot_q, cg_alpha);
        // x <- x + cg_alpha * d
        saxpy<<<num_blocks, BLK_SIZE>>>(cg_alpha, d, x, vec_size);
        // r <- r - cg_alpha * q
        smaxpy<<<num_blocks, BLK_SIZE>>>(cg_alpha, q, r, vec_size);
        scalar_cpy<<<1, 1>>>(r_dot_r_old, r_dot_r);
        if (precondition != -1) {
            // z <- M * r
            spmv_dia<<<num_blocks, BLK_SIZE>>>(M, M_offsets, vec_size, vec_size, m_num_diag, r, z);
            vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, z, r_dot_r, vec_size);
        } else {
            vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, r, r_dot_r, vec_size);
        }
        // cg_beta <- r_dot_r / r_dot_r_old
        scalar_div<<<1, 1>>>(r_dot_r, r_dot_r_old, cg_beta);
        if (precondition != -1) {
            // d <- z + cg_beta * d
            saxpy2<<<num_blocks, BLK_SIZE>>>(cg_beta, z, d, vec_size);
        } else {
            // d <- r + cg_beta *d
            saxpy2<<<num_blocks, BLK_SIZE>>>(cg_beta, r, d, vec_size);
        }
        it++;
        hipMemcpy(&delta_new, r_dot_r, sizeof(Real), hipMemcpyDeviceToHost);
    }
}


void get_abs_max(Real *input, Real *res, Real &out, int size) {
    int num_blks(get_num_blks(size));
    int smemsize = min(BLK_SIZE, size);
    reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(input, res, size);
    while (num_blks != 1) {
        size = ceil(size / Real(BLK_SIZE));
        smemsize = min(BLK_SIZE, size);
        num_blks = get_num_blks(size);
        reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(res, res, size);
    }
    hipMemcpy(&out, res, sizeof(Real), hipMemcpyDeviceToHost);
}

void solve_pcg2(Real *A, int *A_offsets, int num_diag, Real *x, Real *b, Real *q, Real *d, Real *r, Real *rho_old,
               Real *rho, Real *z, Real *cg_beta, Real *residual, Real &delta_new, Real *cg_alpha, Real *d_dot_q, int precondition,
               Real *M, int *M_offsets, int m_num_diag, uint32_t &it, uint32_t max_iter, Real eps, int vec_size) {
    int num_blocks = get_num_blks(vec_size);
    Real residual_out;
    hipMemcpy(r, b, vec_size * sizeof(Real), hipMemcpyDeviceToDevice);
    hipMemset(x, 0, vec_size * sizeof(Real));
    get_abs_max(r, residual, residual_out, vec_size);
    if (residual_out == 0) {
        it = 0;
        return;
    }
    if (precondition != -1) {
        spmv_dia<<<num_blocks, BLK_SIZE>>>(M, M_offsets, vec_size, vec_size, m_num_diag, r, d);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, d, rho, vec_size);
    } else {
        hipMemcpy(d, b, vec_size * sizeof(Real), hipMemcpyDeviceToDevice);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, r, rho, vec_size);
    }
    Real residual_0 = residual_out;
    it = 0;
    while (it < max_iter) {
        // q <- A * d
        hipMemset(q, 0, vec_size * sizeof(Real));
        spmv_dia<<<num_blocks, BLK_SIZE>>>(A, A_offsets, vec_size, vec_size, num_diag, d, q);
        vec_dot_vec<<<num_blocks, BLK_SIZE>>>(d, q, d_dot_q, vec_size);
        // cg_alpha <- rho / d_dot_q
        scalar_div<<<1, 1>>>(rho, d_dot_q, cg_alpha);
        // x <- x - cg_alpha * d
        smaxpy<<<num_blocks, BLK_SIZE>>>(cg_alpha, d, x, vec_size);
        // r <- r - cg_alpha * q
        smaxpy<<<num_blocks, BLK_SIZE>>>(cg_alpha, q, r, vec_size);
        get_abs_max(r, residual, residual_out, vec_size);
        if (residual_out <= eps) {
            break;
        }
        scalar_cpy<<<1, 1>>>(rho_old, rho);
        if (precondition != -1) {
            // z <- M * r
            spmv_dia<<<num_blocks, BLK_SIZE>>>(M, M_offsets, vec_size, vec_size, m_num_diag, r, z);
            vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, z, rho, vec_size);
        } else {
            vec_dot_vec<<<num_blocks, BLK_SIZE>>>(r, r, rho, vec_size);
        }
        // cg_beta <- rho / rho_old
        scalar_div<<<1, 1>>>(rho, rho_old, cg_beta);
        if (precondition != -1) {
            // d <- z + cg_beta * d
            saxpy2<<<num_blocks, BLK_SIZE>>>(cg_beta, z, d, vec_size);
        } else {
            // d <- r + cg_beta *d
            saxpy2<<<num_blocks, BLK_SIZE>>>(cg_beta, r, d, vec_size);
        }
        it++;
    }
    delta_new = residual_out / residual_0;
}

__global__ void sor_iter(Real *P, Real *RS, Real coeff, int *cell_type, int imax, int jmax, Real omega, Real inv_dx,
                         Real inv_dy, int parity) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }

    if (parity == 0 && ((i + j) % 2) == 0) {
        return;
    } else if (parity == 1 && ((i + j) % 2) == 1) {
        return;
    }
    Real p_stencil[4] = {at(P, i + 1, j), at(P, i - 1, j), at(P, i, j + 1), at(P, i, j - 1)};
    at(P, i, j) = (1 - omega) * at(P, i, j) + coeff * (sor_helper(p_stencil, inv_dx, inv_dy) - at(RS, i, j));
}

__global__ void calc_residual(Real *P, Real *RS, int *cell_type, int imax, int jmax, Real inv_dx, Real inv_dy,
                              Real *residual) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }

    Real rloc = 0;
    Real p_laplacian[5] = {at(P, i + 1, j), at(P, i, j), at(P, i - 1, j), at(P, i, j + 1), at(P, i, j - 1)};

    Real val = laplacian_5(p_laplacian, inv_dx, inv_dy) - at(RS, i, j);
    rloc += val * val;
    at(residual, i, j) = rloc;
}

__global__ void reduce_residual(Real *residual, Real *o, int size) {
    __shared__ Real sdata[BLK_SIZE];
    uint32_t tid = threadIdx.x;
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i == 0) {
        *o = 0;
    }
    if (i < size) {
        sdata[tid] = residual[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = sdata[tid] + sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(o, sdata[0]);
    }
}

__global__ void negate_p(Real *p, int size) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        p[i] = -p[i];
    }
}

__device__ Real vel_kernel(Real fg, Real dt, Real p[2], Real inv_dxy) { return fg - dt * inv_dxy * (p[1] - p[0]); }

__global__ void calc_vel(Real *u, Real *v, Real *p, Real *f, Real *g, int *cell_type, Real dt, int imax, int jmax,
                         Real dx, Real dy) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;

    Real p_diff_u[2] = {at(p, i, j), at(p, i + 1, j)};
    Real p_diff_v[2] = {at(p, i, j), at(p, i, j + 1)};
    /*  at(u, i, j) = vel_kernel(at(f, i, j), *dt, p_diff_u, inv_dx);
      at(v, i, j) = vel_kernel(at(g, i, j), *dt, p_diff_v, inv_dy);*/
    at(u, i, j) = at(f, i, j) - dt * inv_dx * (p_diff_u[1] - p_diff_u[0]);
    at(v, i, j) = at(g, i, j) - dt * inv_dy * (p_diff_v[1] - p_diff_v[0]);
}

__global__ void enforce_boundary(Real *u, int *row_start, int *col_idx, Real *mat, Real *rhs_vec, int size) {
    uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < size) {
        Real sum = 0;
        for (int j = row_start[row]; j < row_start[row + 1]; j++) {
            sum += mat[j] * u[col_idx[j]];
        }
        u[row] = sum + 2 * rhs_vec[row];
    }
}

void uv_boundary(Real *u, Real *v, int *row_start_u, int *row_start_v, int *col_idx_u, int *col_idx_v, Real *mat_u,
                 Real *mat_v, Real *rhs_vec_u, Real *rhs_vec_v, int size) {
    int num_blks(get_num_blks(size));
    enforce_boundary<<<num_blks, BLK_SIZE>>>(u, row_start_u, col_idx_u, mat_u, rhs_vec_u, size);
    enforce_boundary<<<num_blks, BLK_SIZE>>>(v, row_start_v, col_idx_v, mat_v, rhs_vec_v, size);
}



void t_boundary(Real *t, int *row_start_t, int *col_idx_t, Real *mat_t, Real *rhs_vec_t, int size) {
    int num_blks(get_num_blks(size));
    enforce_boundary<<<num_blks, BLK_SIZE>>>(t, row_start_t, col_idx_t, mat_t, rhs_vec_t, size);
}

__global__ void calc_t(Real *u, Real *v, Real dx, Real dy, Real *t_new, Real *t_old, int *cell_type, Real alpha,
                       Real dt, Real gamma, int imax, int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    Real u_stencil[2] = {at(u, i - 1, j), at(u, i, j)};
    Real v_stencil[2] = {at(v, i, j - 1), at(v, i, j)};

    Real t_laplacian[5] = {at(t_old, i + 1, j), at(t_old, i, j), at(t_old, i - 1, j), at(t_old, i, j + 1),
                           at(t_old, i, j - 1)};

    at(t_new, i, j) = at(t_new, i, j) + dt * (alpha * laplacian_5(t_laplacian, inv_dx, inv_dy) -
                                              convection_uT(u_stencil, t_laplacian, inv_dx, gamma) -
                                              convection_vT(v_stencil, t_laplacian, inv_dy, gamma));
}

__global__ void calculate_nu_t(Real *NU_T, Real *K, Real *EPS, int *cell_type, Real _nu, int imax, int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real kij = at(K, i, j);
    Real epsij = at(EPS, i, j);
    at(NU_T, i, j) = 0.09 * kij * kij / epsij + _nu;
}

__global__ void calculate_nu_ij(Real *NU_I, Real *NU_J, Real *K, Real *EPS, int *cell_type, Real _nu, int imax,
                                int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real num_i = (at(K, i, j) + at(K, i + 1, j)) / 2;
    Real denom_i = (at(EPS, i, j) + at(EPS, i + 1, j)) / 2;

    Real num_j = (at(K, i, j) + at(K, i, j + 1)) / 2;
    Real denom_j = (at(EPS, i, j) + at(EPS, i, j + 1)) / 2;

    at(NU_I, i, j) = 0.09 * num_i * num_i / denom_i;
    at(NU_J, i, j) = 0.09 * num_j * num_j / denom_j;
}

__global__ void calculate_k_and_epsilon(Real *K_old, Real *EPS_old, Real *K, Real *EPS, Real *NU_T, Real *NU_I,
                                        Real *NU_J, Real *U, Real *V, int *cell_type, Real _nu, int imax, int jmax,
                                        Real dt, Real inv_dx, Real inv_dy) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real f2_coeff = 1;
    auto nut = at(NU_T, i, j);
    auto kij = at(K_old, i, j);
    auto eij = at(EPS_old, i, j);
    Real K_stencil[5] = {at(K_old, i + 1, j), at(K_old, i, j), at(K_old, i - 1, j), at(K_old, i, j + 1),
                         at(K_old, i, j - 1)};
    Real EPS_stencil[5] = {at(EPS_old, i + 1, j), at(EPS_old, i, j), at(EPS_old, i - 1, j), at(EPS_old, i, j + 1),
                           at(EPS_old, i, j - 1)};
    Real U_diff[2] = {at(U, i - 1, j), at(U, i, j)};
    Real V_diff[2] = {at(V, i, j - 1), at(V, i, j)};
    Real NU_I_diff[2] = {at(NU_I, i, j), at(NU_I, i - 1, j)};
    Real NU_J_diff[2] = {at(NU_J, i, j), at(NU_J, i, j - 1)};
    Real U_stencil[6] = {at(U, i, j),         at(U, i - 1, j), at(U, i, j + 1),
                         at(U, i - 1, j + 1), at(U, i, j - 1), at(U, i - 1, j - 1)};
    Real V_stencil[6] = {at(V, i, j),         at(V, i, j - 1), at(V, i + 1, j),
                         at(V, i + 1, j - 1), at(V, i - 1, j), at(V, i - 1, j - 1)};
    auto k1_1 = convecton_uKEPS(U_diff, K_stencil, inv_dx);
    auto k1_2 = convecton_vKEPS(V_diff, K_stencil, inv_dy);
    auto e1_1 = convecton_uKEPS(U_diff, EPS_stencil, inv_dx);
    auto e1_2 = convecton_vKEPS(V_diff, EPS_stencil, inv_dy);

    auto k2 = laplacian_nu(K_stencil, NU_I_diff, NU_J_diff, inv_dx, inv_dy, _nu, 1);
    auto e2 = laplacian_nu(EPS_stencil, NU_I_diff, NU_J_diff, inv_dx, inv_dy, _nu, 1.3);

    auto k3 = nut * mean_strain_rate_squared(U_stencil, V_stencil, inv_dx, inv_dy);
    auto e3 = 1.44 * eij * k3 / kij;
    auto e4 = f2_coeff * 1.92 * eij * eij / kij;
    auto kij_new = kij + dt * (-(k1_1 + k1_2) + k2 + k3 - eij);
    auto epsij_new = eij + dt * (-(e1_1 + e1_2) + e2 + e3 - e4);
    at(K, i, j) = kij_new;
    at(EPS, i, j) = epsij_new;
}

__global__ void calc_fg(Real *f, Real *g, Real *u, Real *v, bool calc_temp, Real dx, Real dy, Real *t, int *cell_type,
                        Real dt, Real gamma, Real nu, Real beta, Real gx, Real gy, int imax, int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    // 5-point + 1 stencil for U and V
    Real u_stencil[6] = {at(u, i + 1, j), at(u, i, j),     at(u, i - 1, j),
                         at(u, i, j + 1), at(u, i, j - 1), at(u, i - 1, j + 1)};
    Real v_stencil[6] = {at(v, i + 1, j), at(v, i, j),     at(v, i - 1, j),
                         at(v, i, j + 1), at(v, i, j - 1), at(v, i + 1, j - 1)};

    // Calculate fluxes
    at(f, i, j) = at(u, i, j) + dt * (nu * laplacian(u_stencil, inv_dx, inv_dy) -
                                      convection_u(u_stencil, v_stencil, inv_dx, inv_dy, gamma));
    at(g, i, j) = at(v, i, j) + dt * (nu * laplacian(v_stencil, inv_dx, inv_dy) -
                                      convection_v(u_stencil, v_stencil, inv_dx, inv_dy, gamma));

    if (calc_temp) {
        Real term1 = at(t, i, j) + at(t, i + 1, j);
        Real term2 = at(t, i, j) + at(t, i, j + 1);
        at(f, i, j) -= beta * dt / 2 * (term1)*gx;
        at(g, i, j) -= beta * dt / 2 * (term2)*gy;
    }
}

__global__ void calc_fg_turbulent(Real *f, Real *g, Real *u, Real *v, Real *NU_T, bool calc_temp, Real dx, Real dy,
                                  Real *t, int *cell_type, Real dt, Real gamma, Real nu, Real beta, Real gx, Real gy,
                                  int imax, int jmax) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    Real inv_dx2 = inv_dx * inv_dx;
    Real inv_dy2 = inv_dy * inv_dy;
    // 5-point + 1 stencil for U and V
    Real u_stencil[6] = {at(u, i + 1, j), at(u, i, j),     at(u, i - 1, j),
                         at(u, i, j + 1), at(u, i, j - 1), at(u, i - 1, j + 1)};
    Real v_stencil[6] = {at(v, i + 1, j), at(v, i, j),     at(v, i - 1, j),
                         at(v, i, j + 1), at(v, i, j - 1), at(v, i + 1, j - 1)};

    Real nu_term1 = (at(NU_T, i, j) + at(NU_T, i + 1, j)) / 2;
    Real nu_term2 = (at(NU_T, i, j) + at(NU_T, i, j + 1)) / 2;
    // Calculate fluxes
    at(f, i, j) = at(u, i, j) + dt * (nu_term1 * laplacian(u_stencil, inv_dx, inv_dy) -
                                      convection_u(u_stencil, v_stencil, inv_dx, inv_dy, gamma));
    at(g, i, j) = at(v, i, j) + dt * (nu_term2 * laplacian(v_stencil, inv_dx, inv_dy) -
                                      convection_v(u_stencil, v_stencil, inv_dx, inv_dy, gamma));

    if (calc_temp) {
        Real term1 = at(t, i, j) + at(t, i + 1, j);
        Real term2 = at(t, i, j) + at(t, i, j + 1);
        at(f, i, j) -= beta * dt / 2 * (term1)*gx;
        at(g, i, j) -= beta * dt / 2 * (term2)*gy;
    }
}

__global__ void fg_boundary(Real *f, Real *g, Real *u, Real *v, int imax, int jmax, uint32_t *neighborhood,
                            int *cell_type) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 1) {
        return;
    }
    uint32_t type = at(neighborhood, i, j) >> 8;
    uint32_t neighbors = at(neighborhood, i, j) & 0xFF;

    if ((neighbors & 0x1) == 1) {
        at(f, i, j) = at(u, i, j);
    }
    if ((neighbors & 0x2) == 2) {
        at(f, i - 1, j) = at(u, i - 1, j);
    }
    if ((neighbors & 0x4) == 4) {
        at(g, i, j) = at(v, i, j);
    }
    if ((neighbors & 0x8) == 8) {
        at(g, i, j - 1) = at(v, i, j - 1);
    }
}

__global__ void p_boundary(Real *p, int imax, int jmax, uint32_t *neighborhood, int *cell_type, Real PI) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 1) {
        return;
    }
    uint32_t type = at(neighborhood, i, j) >> 8;
    uint32_t neighbors = at(neighborhood, i, j) & 0xFF;
    if (type == 0) { // Outlet
        at(p, i, j) = PI;
    } else {
        int diag = 0;
        if ((neighbors & 0x10) == 16) { // Right + top
            diag = 1;
            at(p, i, j) = (at(p, i + 1, j) + at(p, i, j + 1)) / 2;
        }
        if ((neighbors & 0x20) == 32) { // Right + bottom
            diag = 1;
            at(p, i, j) = (at(p, i + 1, j) + at(p, i, j - 1)) / 2;
        }
        if ((neighbors & 0x40) == 64) { // Left + top
            diag = 1;
            at(p, i, j) = (at(p, i - 1, j) + at(p, i, j + 1)) / 2;
        }
        if ((neighbors & 0x80) == 128) { // Left + bottom
            diag = 1;
            at(p, i, j) = (at(p, i - 1, j) + at(p, i, j - 1)) / 2;
        }
        if (!diag) {
            if ((neighbors & 0x1) == 1) { // Right
                at(p, i, j) = at(p, i + 1, j);
            }
            if ((neighbors & 0x2) == 2) { // Left
                at(p, i, j) = at(p, i - 1, j);
            }
            if ((neighbors & 0x4) == 4) { // Top
                at(p, i, j) = at(p, i, j + 1);
            }
            if ((neighbors & 0x8) == 8) { // Bottom
                at(p, i, j) = at(p, i, j - 1);
            }
        }
    }
}

__global__ void nu_t_boundary(Real *NU_T, Real *K, Real *EPS, int imax, int jmax, uint32_t *neighborhood,
                              int *cell_type, Real wk, Real weps, Real _nu) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);
    if (i >= imax || j >= jmax || is_fluid == 1) {
        return;
    }
    uint32_t type = at(neighborhood, i, j) >> 8;
    uint32_t neighbors = at(neighborhood, i, j) & 0xFF;
    Real k = 0;
    Real eps = 0;
    if (type == 1) { // Inlet
        int valid = 0;
        if ((neighbors & 0x1) == 1) { // Right
            k = 2 * wk - at(K, i + 1, j);
            eps = 2 * weps - at(EPS, i + 1, j);
            valid = 1;
        }
        if ((neighbors & 0x2) == 2) { // Left
            k = 2 * wk - at(K, i - 1, j);
            eps = 2 * weps - at(EPS, i - 1, j);
            valid = 1;
        }
        if ((neighbors & 0x4) == 4) { // Top
            k = 2 * wk - at(K, i, j + 1);
            eps = 2 * weps - at(EPS, i, j + 1);
            valid = 1;
        }
        if ((neighbors & 0x8) == 8) { // Bottom
            k = 2 * wk - at(K, i, j - 1);
            eps = 2 * weps - at(EPS, i, j - 1);
            valid = 1;
        }
        if (valid) {
            at(K, i, j) = k;
            at(EPS, i, j) = eps;
            at(NU_T, i, j) = 0.09 * k * k / eps + _nu;
        }
    } else { // Other
        int diag = 0;
        int valid = 0;
        if ((neighbors & 0x10) == 16) { // Right + top
            diag = 1;
            k = (at(K, i + 1, j) + at(K, i, j + 1)) / 2;
            eps = (at(EPS, i + 1, j) + at(EPS, i, j + 1)) / 2;
        }
        if ((neighbors & 0x20) == 32) { // Right + bottom
            diag = 1;
            k = (at(K, i + 1, j) + at(K, i, j - 1)) / 2;
            eps = (at(EPS, i + 1, j) + at(EPS, i, j - 1)) / 2;
        }
        if ((neighbors & 0x40) == 64) { // Left + top
            diag = 1;
            k = (at(K, i - 1, j) + at(K, i, j + 1)) / 2;
            eps = (at(EPS, i - 1, j) + at(EPS, i, j + 1)) / 2;
        }
        if ((neighbors & 0x80) == 128) { // Left + bottom
            diag = 1;
            k = (at(K, i - 1, j) + at(K, i, j - 1)) / 2;
            eps = (at(EPS, i - 1, j) + at(EPS, i, j - 1)) / 2;
        }
        if (!diag) {
            if ((neighbors & 0x1) == 1) { // Right
                k = at(K, i + 1, j);
                eps = at(EPS, i + 1, j);
                valid = 1;
            }
            if ((neighbors & 0x2) == 2) { // Left
                k = at(K, i - 1, j);
                eps = at(EPS, i - 1, j);
                valid = 1;
            }
            if ((neighbors & 0x4) == 4) { // Top
                k = at(K, i, j + 1);
                eps = at(EPS, i, j + 1);
                valid = 1;
            }
            if ((neighbors & 0x8) == 8) { // Bottom
                k = at(K, i, j - 1);
                eps = at(EPS, i, j - 1);
                valid = 1;
            }
        }
        if (diag || valid) {
            at(K, i, j) = k;
            at(EPS, i, j) = eps;
            at(NU_T, i, j) = 0.09 * k * k / eps + _nu;
        }
    }
}

void solve_sor(Real *P, Real *P_tmp, Real *P_residual, Real *P_residual_out, uint32_t *neighborhood, int imax, int jmax,
               Real *RS, int *cell_type, uint32_t &it, uint32_t max_iter, Real dx, Real dy, Real PI, Real tolerance,
               Real &res, int num_fluid_cells) {
    it = 0;
    const Real omega = 1.7;
    auto grid_size = imax * jmax;
    Real coeff = omega / (2 * (1 / (dx * dx) + 1 / (dy * dy)));
    dim3 blk_size_2d(BLK_SIZE_2D, BLK_SIZE_2D);
    dim3 num_blks_2d = get_num_blks_2d(imax, jmax);
    int num_blks_1d(get_num_blks(grid_size));
    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    res = REAL_MAX;
    while (it < max_iter && res > tolerance) {
        sor_iter<<<num_blks_2d, blk_size_2d>>>(P, RS, coeff, cell_type, imax, jmax, omega, inv_dx, inv_dy, 0);
        sor_iter<<<num_blks_2d, blk_size_2d>>>(P, RS, coeff, cell_type, imax, jmax, omega, inv_dx, inv_dy, 1);
        hipMemset(P_residual, 0, grid_size * sizeof(Real));
        calc_residual<<<num_blks_2d, blk_size_2d>>>(P, RS, cell_type, imax, jmax, inv_dx, inv_dy, P_residual);
        reduce_residual<<<num_blks_1d, BLK_SIZE>>>(P_residual, P_residual_out, grid_size);
        p_boundary<<<num_blks_2d, blk_size_2d>>>(P, imax, jmax, neighborhood, cell_type, PI);
        hipMemcpy(&res, P_residual_out, sizeof(Real), hipMemcpyDeviceToHost);
        res = std::sqrt(res / num_fluid_cells);
        it++;
    }
}

__global__ void calc_rs(Real *f, Real *g, Real *rs, Real dx, Real dy, int imax, int jmax, Real dt, int *cell_type) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;
    int is_fluid = at(cell_type, i, j);

    if (i >= imax || j >= jmax || is_fluid == 0) {
        return;
    }

    Real inv_dx = 1 / dx;
    Real inv_dy = 1 / dy;
    Real f_diff[2] = {at(f, i, j), at(f, i - 1, j)};
    Real g_diff[2] = {at(g, i, j), at(g, i, j - 1)};

    Real df = inv_dx * (f_diff[0] - f_diff[1]);
    Real dg = inv_dy * (g_diff[0] - g_diff[1]);
    at(rs, i, j) = (df + dg) * 1 / dt;
}

Real calculate_dt(int imax, int jmax, Real *u, Real *v, Real *u_residual, Real *v_residual, Real *nu_residual,
                  Real *k_residual, Real *eps_residual, Real *nu_t, Real *k, Real *eps, Real dx, Real dy, Real tau,
                  Real nu, Real alpha, bool calc_temp, bool turbulent) {
    // Calculate uv max
    int size = imax * jmax;
    int num_blks(get_num_blks(size));
    Real u_max_abs = 0;
    Real v_max_abs = 0;
    Real nu_min = REAL_MAX;
    Real k_max;
    Real eps_max;

    Real dx2 = dx * dx;
    Real dy2 = dy * dy;
    int smemsize = min(BLK_SIZE, size);
    std::vector<Real> ucpu(imax * jmax);
    std::vector<Real> vcpu(imax * jmax);
    std::vector<Real> ures(imax * jmax);
    std::vector<Real> vres(imax * jmax);

    reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(u, u_residual, size);
    reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(v, v_residual, size);
    if (turbulent) {
        reduce_min<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(nu_t, nu_residual, size);
        reduce_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(k, k_residual, size);
        reduce_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(eps, eps_residual, size);
    }
    while (num_blks != 1) {
        size = ceil(size / Real(BLK_SIZE));
        smemsize = min(BLK_SIZE, size);
        num_blks = get_num_blks(size);
        reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(u_residual, u_residual, size);
        reduce_abs_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(v_residual, v_residual, size);
        if (turbulent) {
            reduce_min<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(nu_t, nu_residual, size);
            reduce_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(k, k_residual, size);
            reduce_max<<<num_blks, BLK_SIZE, smemsize * sizeof(Real)>>>(eps, eps_residual, size);
        }
    }
    hipMemcpy(&u_max_abs, u_residual, sizeof(Real), hipMemcpyDeviceToHost);
    hipMemcpy(&v_max_abs, v_residual, sizeof(Real), hipMemcpyDeviceToHost);
    if (turbulent) {
        hipMemcpy(&nu_min, nu_residual, sizeof(Real), hipMemcpyDeviceToHost);
        hipMemcpy(&k_max, k_residual, sizeof(Real), hipMemcpyDeviceToHost);
        hipMemcpy(&eps_max, eps_residual, sizeof(Real), hipMemcpyDeviceToHost);
    }
    Real min_cond = std::min(dx / u_max_abs, dy / v_max_abs);

    nu_min = (nu_min == REAL_MAX || nu_min == 0) ? nu : nu_min;

    if (nu_min != 0) {
        Real cond_spatial = 1.0 / (2.0 * nu) * ((dx2 * dy2) / (dx2 + dy2));
        min_cond = std::min(min_cond, cond_spatial);
    }

    if (calc_temp) {
        Real inv_dx = 1 / dx;
        Real inv_dx2 = inv_dx * inv_dx;
        Real inv_dy = 1 / dy;
        Real inv_dy2 = inv_dy * inv_dy;
        Real cond_temp = 1 / (2 * alpha * (inv_dx2 + inv_dy2));
        min_cond = std::min(min_cond, cond_temp);
    }
    if (turbulent) {
        Real cond_5 = 1 / (2 * k_max * (1 / dx2 + 1 / dy2));
        Real cond_6 = 1 / (2 * eps_max * (1 / dx2 + 1 / dy2));
        min_cond = std::min(min_cond, cond_5);
        min_cond = std::min(min_cond, cond_6);
    }

    return tau * min_cond;
}
void CudaSolver::initialize() {
    auto grid_x = _grid.imaxb();
    auto grid_y = _grid.jmaxb();
    auto grid_size = grid_x * grid_y;
    build_pcg_matrix(_field, _grid, _boundaries, A_pcg, U_pcg, V_pcg, T_pcg, U_RHS, V_RHS, T_RHS, U_fixed, V_fixed,
                     T_fixed);
    // Preprocess
    std::vector<int> is_fluid(_grid.imaxb() * _grid.jmaxb(), 0);
    for (const auto &current_cell : _grid.fluid_cells()) {
        int i = current_cell->i();
        int j = current_cell->j();
        is_fluid[_grid.imaxb() * j + i] = 1;
    }

    std::vector<BoundaryData> neighbors(_grid.imaxb() * _grid.jmaxb());
    for (const auto &boundary : _boundaries) {
        uint32_t type = boundary->get_type();
        auto cells = boundary->_cells;
        for (auto &cell : *cells) {
            int i = cell->i();
            int j = cell->j();
            BoundaryData data;
            uint32_t type = boundary->get_type();
            data.neighborhood |= type << 8;
            // data.idx = _grid.imaxb() * j + i;
            if (cell->is_border(border_position::RIGHT)) {
                data.neighborhood |= 1;
            }
            if (cell->is_border(border_position::LEFT)) {
                data.neighborhood |= 2;
            }
            if (cell->is_border(border_position::TOP)) {
                data.neighborhood |= 4;
            }
            if (cell->is_border(border_position::BOTTOM)) {
                data.neighborhood |= 8;
            }
            if (cell->is_border(border_position::RIGHT) && cell->is_border(border_position::TOP)) {
                data.neighborhood |= 16;
            }
            if (cell->is_border(border_position::RIGHT) && cell->is_border(border_position::BOTTOM)) {
                data.neighborhood |= 32;
            }
            if (cell->is_border(border_position::LEFT) && cell->is_border(border_position::TOP)) {
                data.neighborhood |= 64;
            }
            if (cell->is_border(border_position::LEFT) && cell->is_border(border_position::BOTTOM)) {
                data.neighborhood |= 128;
            }
            neighbors[j * _grid.imaxb() + i] = data;
        }
    }
    DiagonalSparseMatrix<Real> A_matrix_diag =
        create_diagonal_matrix(A_pcg, _grid.imaxb(), _grid.jmaxb(), {-_grid.imaxb(), -1, 0, 1, _grid.imaxb()});
    DiagonalSparseMatrix<Real> A_precond_diag;
    if (_preconditioner != -1) {
        A_precond_diag = create_preconditioner_spai(A_pcg, _grid, _preconditioner);
    }
    num_offsets_a = A_matrix_diag.offsets.size();
    num_offsets_m = A_precond_diag.offsets.size();
    auto t_matrix_data = T_fixed.value.data();
    auto t_matrix_size = T_fixed.value.size();
    auto t_row_start_data = T_fixed.rowstart.data();
    auto t_row_start_size = T_fixed.rowstart.size();
    auto t_col_idx_data = T_fixed.colindex.data();
    auto t_col_idx_size = T_fixed.colindex.size();
    auto t_rhs_data = T_RHS.data();
    auto t_rhs_size = T_RHS.size();
    auto u_matrix_data = U_fixed.value.data();
    auto u_matrix_size = U_fixed.value.size();
    auto v_matrix_data = V_fixed.value.data();
    auto v_matrix_size = V_fixed.value.size();
    auto u_row_start_data = U_fixed.rowstart.data();
    auto u_row_start_size = U_fixed.rowstart.size();
    auto u_col_idx_data = U_fixed.colindex.data();
    auto u_col_idx_size = U_fixed.colindex.size();
    auto v_row_start_data = V_fixed.rowstart.data();
    auto v_row_start_size = V_fixed.rowstart.size();
    auto v_col_idx_data = V_fixed.colindex.data();
    auto v_col_idx_size = V_fixed.colindex.size();
    auto u_rhs_data = U_RHS.data();
    auto u_rhs_size = U_RHS.size();
    auto v_rhs_data = V_RHS.data();
    auto v_rhs_size = V_RHS.size();

    hipMalloc(&U, grid_size * sizeof(Real));
    hipMalloc(&V, grid_size * sizeof(Real));
    hipMalloc(&F, grid_size * sizeof(Real));
    hipMalloc(&G, grid_size * sizeof(Real));
    hipMalloc(&P, grid_size * sizeof(Real));
    hipMalloc(&P_temp, grid_size * sizeof(Real));

    hipMalloc(&RS, grid_size * sizeof(Real));
    hipMalloc(&U_residual, grid_size * sizeof(Real));
    hipMalloc(&V_residual, grid_size * sizeof(Real));
    if (_turb_model != 0) {
        hipMalloc(&NU_residual, grid_size * sizeof(Real));
        hipMalloc(&K_residual, grid_size * sizeof(Real));
        hipMalloc(&EPS_residual, grid_size * sizeof(Real));
        hipMalloc(&NU_T, grid_size * sizeof(Real));
        hipMalloc(&NU_I, grid_size * sizeof(Real));
        hipMalloc(&NU_J, grid_size * sizeof(Real));
        hipMalloc(&K, grid_size * sizeof(Real));
        hipMalloc(&K_old, grid_size * sizeof(Real));
        hipMalloc(&EPS, grid_size * sizeof(Real));
        hipMalloc(&EPS_old, grid_size * sizeof(Real));

        hipMemset(NU_residual, 0, grid_size * sizeof(Real));
        hipMemset(K_residual, 0, grid_size * sizeof(Real));
        hipMemset(EPS_residual, 0, grid_size * sizeof(Real));
        hipMemset(NU_T, 0, grid_size * sizeof(Real));
        hipMemset(NU_I, 0, grid_size * sizeof(Real));
        hipMemset(NU_J, 0, grid_size * sizeof(Real));
        hipMemcpy(K, _field._K._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice);
        hipMemcpy(EPS, _field._EPS._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice);
    }
    hipMalloc(&P_residual, grid_size * sizeof(Real));
    hipMalloc(&cell_type, grid_size * sizeof(int));
    hipMalloc(&row_start_u, u_row_start_size * sizeof(int));
    hipMalloc(&row_start_v, v_row_start_size * sizeof(int));
    hipMalloc(&row_start_t, t_row_start_size * sizeof(int));
    hipMalloc(&col_idx_u, u_col_idx_size * sizeof(int));
    hipMalloc(&col_idx_v, v_col_idx_size * sizeof(int));
    hipMalloc(&mat_u, u_matrix_size * sizeof(Real));
    hipMalloc(&mat_v, v_matrix_size * sizeof(Real));
    hipMalloc(&rhs_vec_u, u_rhs_size * sizeof(Real));
    hipMalloc(&rhs_vec_v, v_rhs_size * sizeof(Real));
    hipMalloc(&neighborhood, neighbors.size() * sizeof(uint32_t));

    hipMalloc(&A, A_matrix_diag.data.size() * sizeof(Real));
    hipMalloc(&A_offsets, A_matrix_diag.offsets.size() * sizeof(uint32_t));
    if (_preconditioner != -1) {
        hipMalloc(&M, A_precond_diag.data.size() * sizeof(Real));
        hipMalloc(&M_offsets, A_precond_diag.offsets.size() * sizeof(uint32_t));
    }
    hipMalloc(&q, grid_size * sizeof(Real));
    hipMalloc(&d, grid_size * sizeof(Real));
    hipMalloc(&r, grid_size * sizeof(Real));
    hipMalloc(&z, grid_size * sizeof(Real));
    hipMalloc(&r_dot_r, sizeof(Real));
    hipMalloc(&r_dot_r_old, sizeof(Real));
    hipMalloc(&d_dot_q, sizeof(Real));
    hipMalloc(&p_residual_out, sizeof(Real));
    hipMalloc(&cg_alpha, sizeof(Real));
    hipMalloc(&cg_beta, sizeof(Real));

    chk(hipMemcpy(U, _field._U._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(V, _field._V._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(P, _field._P._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(mat_u, u_matrix_data, u_matrix_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(row_start_u, u_row_start_data, u_row_start_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(col_idx_u, u_col_idx_data, u_col_idx_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(rhs_vec_u, u_rhs_data, u_rhs_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(mat_v, v_matrix_data, v_matrix_size * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(row_start_v, v_row_start_data, v_row_start_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(col_idx_v, v_col_idx_data, v_col_idx_size * sizeof(int), hipMemcpyHostToDevice));
    chk(hipMemcpy(rhs_vec_v, v_rhs_data, v_rhs_size * sizeof(Real), hipMemcpyHostToDevice));

    chk(hipMemcpy(neighborhood, neighbors.data(), neighbors.size() * sizeof(uint32_t), hipMemcpyHostToDevice));

    chk(hipMemcpy(A, A_matrix_diag.data.data(), A_matrix_diag.data.size() * sizeof(Real), hipMemcpyHostToDevice));
    chk(hipMemcpy(A_offsets, A_matrix_diag.offsets.data(), A_matrix_diag.offsets.size() * sizeof(int),
                   hipMemcpyHostToDevice));
    if (_field.calc_temp) {
        hipMalloc(&T, grid_size * sizeof(Real));
        hipMalloc(&T_temp, grid_size * sizeof(Real));
        hipMalloc(&mat_t, t_matrix_size * sizeof(Real));
        hipMalloc(&rhs_vec_t, t_rhs_size * sizeof(Real));
        hipMalloc(&col_idx_t, t_col_idx_size * sizeof(int));

        hipMemcpy(mat_t, t_matrix_data, t_matrix_size * sizeof(Real), hipMemcpyHostToDevice);
        hipMemcpy(row_start_t, t_row_start_data, t_row_start_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(col_idx_t, t_col_idx_data, t_col_idx_size * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(rhs_vec_t, t_rhs_data, t_rhs_size * sizeof(Real), hipMemcpyHostToDevice);
        chk(hipMemcpy(T, _field._T._container.data(), grid_size * sizeof(Real), hipMemcpyHostToDevice));
    }

    if (_preconditioner != -1) {
        chk(hipMemcpy(M, A_precond_diag.data.data(), A_precond_diag.data.size() * sizeof(Real),
                       hipMemcpyHostToDevice));
        chk(hipMemcpy(M_offsets, A_precond_diag.offsets.data(), A_precond_diag.offsets.size() * sizeof(int),
                       hipMemcpyHostToDevice));
    }
    chk(hipMemcpy(cell_type, is_fluid.data(), is_fluid.size() * sizeof(int), hipMemcpyHostToDevice));
}
void CudaSolver::solve_pre_pressure(Real &dt) {
    auto grid_x = _grid.imaxb();
    auto grid_y = _grid.jmaxb();
    auto grid_size = grid_x * grid_y;
    dim3 num_blks_1d(get_num_blks(grid_size));
    dim3 blk_size_2d(BLK_SIZE_2D, BLK_SIZE_2D);
    dim3 num_blks_2d = get_num_blks_2d(grid_x, grid_y);
    dt = calculate_dt(_grid.imaxb(), _grid.jmaxb(), U, V, U_residual, V_residual, NU_residual, K_residual, EPS_residual,
                      NU_T, K, EPS, _grid.dx(), _grid.dy(), _field._tau, _field._nu, _field._alpha, _field.calc_temp,
                      _turb_model != 0);
    _field._dt = dt;
    uv_boundary(U, V, row_start_u, row_start_v, col_idx_u, col_idx_v, mat_u, mat_v, rhs_vec_u, rhs_vec_v, grid_size);
    if (_field.calc_temp) {
        t_boundary(T, row_start_t, col_idx_t, mat_t, rhs_vec_t, grid_size);
        chk(hipMemcpy(T_temp, T, grid_size * sizeof(Real), hipMemcpyDeviceToDevice));
        calc_t<<<num_blks_2d, blk_size_2d>>>(U, V, _grid.dx(), _grid.dy(), T, T_temp, cell_type, _field._alpha, dt,
                                             _discretization._gamma, _grid.imaxb(), _grid.jmaxb());
    }
    std::vector<Real> fcpu(grid_size);
    std::vector<Real> gcpu(grid_size);
    if (_turb_model != 0) {
        calc_fg_turbulent<<<num_blks_2d, blk_size_2d>>>(F, G, U, V, NU_T, _field.calc_temp, _grid.dx(), _grid.dy(), T,
                                                        cell_type, dt, _discretization._gamma, _field._nu, _field._beta,
                                                        _field._gx, _field._gy, grid_x, grid_y);
    } else {
        calc_fg<<<num_blks_2d, blk_size_2d>>>(F, G, U, V, _field.calc_temp, _grid.dx(), _grid.dy(), T, cell_type, dt,
                                              _discretization._gamma, _field._nu, _field._beta, _field._gx, _field._gy,
                                              grid_x, grid_y);
    }
    fg_boundary<<<num_blks_2d, blk_size_2d>>>(F, G, U, V, grid_x, grid_y, neighborhood, cell_type);
    calc_rs<<<num_blks_2d, blk_size_2d>>>(F, G, RS, _grid.dx(), _grid.dy(), grid_x, grid_y, dt, cell_type);
}

void CudaSolver::solve_pressure(Real &res, uint32_t &it) {
    if (solver_type == SolverType::PCG) {
        auto grid_x = _grid.imaxb();
        auto grid_y = _grid.jmaxb();
        auto grid_size = grid_x * grid_y;
        int num_blks(get_num_blks(grid_size));
        solve_pcg2(A, A_offsets, num_offsets_a, P, RS, q, d, r, r_dot_r_old, r_dot_r, z, cg_beta, U_residual, res, cg_alpha, d_dot_q,
                  _preconditioner, M, M_offsets, num_offsets_m, it, _max_iter, _tolerance,
                  _grid.imaxb() * _grid.jmaxb());
    } else if (solver_type == SolverType::SOR) {
        solve_sor(P, P_temp, P_residual, p_residual_out, neighborhood, _grid.imaxb(), _grid.jmaxb(), RS, cell_type, it,
                  _max_iter, _grid.dx(), _grid.dy(), _field._PI, _tolerance, res, _grid.fluid_cells().size());
    }
}

void CudaSolver::solve_post_pressure() {
    auto grid_x = _grid.imaxb();
    auto grid_y = _grid.jmaxb();
    auto grid_size = grid_x * grid_y;
    int num_blks(get_num_blks(grid_size));
    dim3 blk_size_2d(BLK_SIZE_2D, BLK_SIZE_2D);
    dim3 num_blks_2d = get_num_blks_2d(grid_x, grid_y);
    calc_vel<<<num_blks_2d, blk_size_2d>>>(U, V, P, F, G, cell_type, _field._dt, grid_x, grid_y, _grid.dx(),
                                           _grid.dy());
    if (_turb_model == 1) {
        chk(hipMemcpy(K_old, K, grid_size * sizeof(Real), hipMemcpyDeviceToDevice));
        chk(hipMemcpy(EPS_old, EPS, grid_size * sizeof(Real), hipMemcpyDeviceToDevice));
        calculate_nu_t<<<num_blks_2d, blk_size_2d>>>(NU_T, K, EPS, cell_type, _field._nu, grid_x, grid_y);
        calculate_nu_ij<<<num_blks_2d, blk_size_2d>>>(NU_I, NU_J, K, EPS, cell_type, _field._nu, grid_x, grid_y);
        calculate_k_and_epsilon<<<num_blks_2d, blk_size_2d>>>(K_old, EPS_old, K, EPS, NU_T, NU_I, NU_J, U, V, cell_type,
                                                              _field._nu, grid_x, grid_y, _field._dt, 1 / _grid.dx(),
                                                              1 / _grid.dy());
        // TODO : Implement KIN and EPSIN
        nu_t_boundary<<<num_blks_2d, blk_size_2d>>>(NU_T, K, EPS, grid_x, grid_y, neighborhood, cell_type, 0, 0,
                                                    _field._nu);
    }
    chk(hipMemcpy(_field._U._container.data(), U, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    chk(hipMemcpy(_field._V._container.data(), V, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    chk(hipMemcpy(_field._P._container.data(), P, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    if (_field.calc_temp) {
        chk(hipMemcpy(_field._T._container.data(), T, grid_size * sizeof(Real), hipMemcpyDeviceToHost));
    }
}

CudaSolver::~CudaSolver() {
    hipFree(U);
    hipFree(V);
    hipFree(F);
    hipFree(G);
    hipFree(P);
    hipFree(T);
    hipFree(T_temp);
    hipFree(RS);
    hipFree(U_residual);
    hipFree(V_residual);
    if (_turb_model != 0) {
        hipFree(NU_residual);
        hipFree(K_residual);
        hipFree(EPS_residual);
        hipFree(NU_T);
        hipFree(NU_I);
        hipFree(NU_J);
        hipFree(K);
        hipFree(K_old);
        hipFree(EPS_old);
        hipFree(EPS);
    }
    hipFree(P_residual);
    hipFree(cell_type);
    hipFree(P);
    hipFree(P_temp);
    hipFree(row_start_u);
    hipFree(row_start_v);
    hipFree(row_start_t);
    hipFree(col_idx_u);
    hipFree(col_idx_v);
    hipFree(col_idx_t);
    hipFree(mat_u);
    hipFree(mat_v);
    hipFree(mat_t);
    hipFree(rhs_vec_u);
    hipFree(rhs_vec_v);
    hipFree(rhs_vec_t);
    hipFree(neighborhood);
    hipFree(A);
    hipFree(A_offsets);
    if (_preconditioner != -1) {
        hipFree(M);
        hipFree(M_offsets);
    }
    hipFree(q);
    hipFree(d);
    hipFree(r);
    hipFree(z);
    hipFree(r_dot_r);
    hipFree(r_dot_r_old);
    hipFree(d_dot_q);
    hipFree(p_residual_out);
    hipFree(cg_alpha);
    hipFree(cg_beta);
}
